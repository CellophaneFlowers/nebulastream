//
// Created by Toso, Lorenzo on 2019-03-28.
//

#include "ReadWriteGPUOperator.h"

void ReadWriteGPUOperator::start_gpu() {
    can_start_gpu = true;
}


void ReadWriteGPUOperator::receive_tuples_from(size_t target_rank, const std::function<void(const StructuredTupleBuffer &, std::atomic_char*)> &tuple_processor) {

    TRACE2("Started routine to receive tuples from %lu!\n", target_rank);

    std::vector<void*> recv_buffer_memory(WRITE_RECEIVE_BUFFER_COUNT);
    std::vector<Buffer*> recv_buffers(WRITE_RECEIVE_BUFFER_COUNT);
    std::vector<RegionToken*> region_tokens(WRITE_RECEIVE_BUFFER_COUNT + 1);
    std::vector<std::shared_ptr<std::thread>> buffer_threads(WRITE_RECEIVE_BUFFER_COUNT, nullptr);

    std::vector<std::atomic_char> buffer_ready_sign(WRITE_RECEIVE_BUFFER_COUNT);
    for (auto &b : buffer_ready_sign)
        b = BUFFER_READY_FLAG;

    auto sign_buffer = connections[target_rank].register_buffer(buffer_ready_sign.data(), WRITE_RECEIVE_BUFFER_COUNT);


    while (!(can_start_gpu && can_start))
    {
        std::this_thread::yield();
    }
    auto start_time = TimeTools::now();

    TRACE2("Allocating GPU memory\n");

    for (size_t i = 0; i <= WRITE_RECEIVE_BUFFER_COUNT; i++)
    {
        if (i < WRITE_RECEIVE_BUFFER_COUNT)
        {
            hipMalloc(&recv_buffer_memory[i], JOIN_WRITE_BUFFER_SIZE);

            recv_buffers[i] = connections[target_rank].register_buffer(recv_buffer_memory[i], JOIN_WRITE_BUFFER_SIZE);
            region_tokens[i] = recv_buffers[i]->createRegionToken();

            hipMemcpy((RegionToken*) recv_buffers[0]->getData() + i, region_tokens[i], sizeof(RegionToken), hipMemcpyHostToDevice);

            TRACE2("Done allocating buffer %lu\n", i);
        } else {
            region_tokens[WRITE_RECEIVE_BUFFER_COUNT] = sign_buffer->createRegionToken();

            hipMemcpy((RegionToken*) recv_buffers[0]->getData() + WRITE_RECEIVE_BUFFER_COUNT, region_tokens[WRITE_RECEIVE_BUFFER_COUNT], sizeof(RegionToken), hipMemcpyHostToDevice);
        }
    }
    TRACE2("Done copying last regiontoken\n");


    TRACE2("Starting to send tokens\n");
    connections[target_rank].send_blocking(recv_buffers[0]);
    TRACE2("Done sending tokens\n");

    size_t index = 0;
    while (true) {
        index++;
        index %= WRITE_RECEIVE_BUFFER_COUNT;

        if (buffer_ready_sign[index] == BUFFER_USED_SENDER_DONE)
            break;

        if (buffer_ready_sign[index] == BUFFER_USED_FLAG) {

            TRACE("Received something on buffer %lu!\n", index);
            if(buffer_threads[index] != nullptr)
                buffer_threads[index]->join();

            buffer_ready_sign[index] = BUFFER_BEING_PROCESSED_FLAG;
            buffer_threads[index] = std::make_shared<std::thread>([this,&recv_buffers,index, &buffer_ready_sign, &tuple_processor] {
                tuple_processor(
                        StructuredTupleBuffer(recv_buffers[index]->getData(), JOIN_WRITE_BUFFER_SIZE),
                        &buffer_ready_sign[index]);
            });
        }
    }

    for (index = 0; index < WRITE_RECEIVE_BUFFER_COUNT; index++)
    {
        if(buffer_threads[index] != nullptr)
            buffer_threads[index]->join();
        if (buffer_ready_sign[index] != BUFFER_READY_FLAG) {
            tuple_processor(StructuredTupleBuffer(recv_buffers[index]->getData(), JOIN_WRITE_BUFFER_SIZE),
                &buffer_ready_sign[index]
            );
        }
    }

    done_with_sending[target_rank] = true;

    auto end_time = TimeTools::now();
    measured_network_times[MPIHelper::get_rank()] = end_time - start_time;


    for (auto &token : region_tokens)
        delete token;
    for (auto &buffer : recv_buffers)
        delete buffer;
    for (auto &memory : recv_buffer_memory)
        hipFree(memory);
}