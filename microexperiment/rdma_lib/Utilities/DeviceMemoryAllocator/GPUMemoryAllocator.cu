#include "hip/hip_runtime.h"
//
// Created by Toso, Lorenzo on 2019-01-08.
//

#include "GPUMemoryAllocator.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "Debug.h"
#include <chrono>

void * GPUMemoryAllocator::allocate_on_device(size_t size) {
    TRACE("Allocating buffer of size %lu on GPU.\n", size);
    void* local_buffer;
    if( hipMalloc(&local_buffer,size) != hipSuccess){
        fprintf(stderr, "ERROR ALLOCATING CUDA-MEMORY!!!\n");
    }

    return local_buffer;
}


void GPUMemoryAllocator::delete_on_device(void* buffer) const {
    TRACE("Allocating buffer on GPU.\n");
    hipFree(buffer);
}

void GPUMemoryAllocator::copy_to_device(void* local, void* device, size_t size) const {
    TRACE("Copying memory to GPU.\n");
    hipMemcpy(device, local, size, hipMemcpyHostToDevice);
}
void GPUMemoryAllocator::copy_from_device(void* local, void* device, size_t size) const {
    TRACE("Copying memory from GPU.\n");
    hipMemcpy(local, device, size, hipMemcpyDeviceToHost);
}

void GPUMemoryAllocator::set_on_device(char value, void* device, size_t size) const {
    hipMemset(device, value, size);
}

void GPUMemoryAllocator::inc_on_device(void* device, size_t size) const {
    char * temp = static_cast<char*>(malloc(size));
    copy_from_device(temp, device, size);
    for (size_t i = 0; i < size; i++){
        temp[i]++;
    }
    copy_to_device(temp, device, size);
    free(temp);
}

void GPUMemoryAllocator::dec_on_device(void* device, size_t size) const {
    char * temp = static_cast<char*>(malloc(size));
    copy_from_device(temp, device, size);
    for (size_t i = 0; i < size; i++){
        temp[i]--;
    }
    copy_to_device(temp, device, size);
    free(temp);
}


__global__
void gpu_wait_for_value(volatile char* memory, volatile char expected_value){
    while(memory[0] != expected_value);
}
__global__
void gpu_wait_for_value_and_reset(volatile char* memory, volatile char expected_value, volatile char reset_value){
    while(memory[0] != expected_value);
    memory[0] = reset_value;
}


void GPUMemoryAllocator::wait_for_value(char* memory, char expected_value) const {
    gpu_wait_for_value<<<1,1>>>(memory, expected_value);
    hipDeviceSynchronize();
}

void GPUMemoryAllocator::wait_for_value_and_reset(char* memory, char expected_value, char reset_value) const {
    gpu_wait_for_value_and_reset<<<1,1>>>(memory, expected_value, reset_value);
    hipDeviceSynchronize();
}

void GPUMemoryAllocator::copy_on_device(void* dest, void* source, size_t size) {
    hipMemcpy(dest, source, size, hipMemcpyDeviceToDevice);
}

bool GPUMemoryAllocator::uses_gpu() const {
    return true;
}


__global__
void add(int size, char * ints, int * sum)
{
    int thread_index = threadIdx.x;
    int block_index = blockIdx.x;
    int block_size = blockDim.x;
    int unique_thread_index = thread_index + block_index * block_size;

    for (int stride = 1; stride <= size/2; stride *= 2) {
        int my_index = 2 * stride * unique_thread_index;

        if(my_index >= size || my_index + stride >= size)
            return;

        ints[my_index] = ints[my_index] + ints[my_index+stride];
    }
    if(thread_index == 0 && block_index == 0)
        *sum = ints[0];
}

/*
int GPUMemoryAllocator::sum(void* device, size_t size) {
    int * sum;
    hipMalloc(&sum, sizeof(int));

    auto start = static_cast<size_t>(std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count());
    add<<<16, 1>>>((int)16, (char*)device, sum);
    hipDeviceSynchronize();
    auto ending = static_cast<size_t>(std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count());

    int cpu_sum;
    hipMemcpy(&cpu_sum, sum, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of %lu values took %lumus\n", size, (ending-start)/1000);
    return cpu_sum;
}
 */