#include "hip/hip_runtime.h"
//
// Created by Toso, Lorenzo on 2019-01-09.
//
#include <sys/time.h>

#include "MPIHelper.h"
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <NumaUtilities.h>
#include <hip/hip_runtime.h>
#include "TimeTools.hpp"
#include "BenchmarkTools.h"
#include "../ComputationParameters.h"
#include "ConnectionInfoProvider/SimpleInfoProvider.h"
#include "VerbsConnection.h"
#include "DeviceMemoryAllocator/DeviceMemoryAllocator.h"
#include "DeviceMemoryAllocator/CPUMemoryAllocator.h"
#include "DeviceMemoryAllocator/GPUMemoryAllocator.h"
#include "Debug.h"

#define SERVER_IP "192.168.5.30"
#define PORT 55355

//#define GPU
#define WRITE
//#define READ
//#define SEND

size_t BUFFER_SIZE = 1;
size_t ITERATIONS = 100;
size_t REPETITIONS = 1000;


#ifdef GPU
__global__
void gpu_wait_for_value_and_reset(volatile char* memory, volatile char expected_value, volatile char reset_value){
    while(memory[0] != expected_value);
    memory[0] = reset_value;
}
#endif


void run_test(const std::string &output_path, size_t numa_node) {
    size_t rank = MPIHelper::get_rank();
    size_t target_rank = rank == 0 ? 1 : 0;

    SimpleInfoProvider info(target_rank, 3-static_cast<u_int16_t>(numa_node * 3), 1, PORT, SERVER_IP);
    VerbsConnection connection(&info);

#ifdef GPU
    char * receive_memory;
    char * send_memory;
    hipMalloc(&receive_memory, BUFFER_SIZE);
    hipMalloc(&send_memory, BUFFER_SIZE);
    hipMemset(send_memory, static_cast<char>(rank+1), BUFFER_SIZE);
    hipMemset(receive_memory, 0, BUFFER_SIZE);
#else
    char * receive_memory = static_cast<char*>(malloc(BUFFER_SIZE));
    char * send_memory = static_cast<char*>(malloc(BUFFER_SIZE));
    memset(receive_memory,0,BUFFER_SIZE);
#endif
    auto receive_buffer = connection.register_buffer(receive_memory, BUFFER_SIZE);
    auto send_buffer = connection.register_buffer(send_memory, BUFFER_SIZE);

    auto remote_receive_token = connection.exchange_region_tokens(receive_buffer);
    RequestToken* pRequestToken = connection.create_request_token();

    connection.barrier();

    std::vector<Timestamp> measured_times;
    printf("Starting Measurement!\n");
    for(size_t iteration = 0; iteration < ITERATIONS; iteration++)
    {
        TRACE("Iteration: %d\n", iteration);


#ifdef SEND
        ReceiveElement receiveElement;
        connection.barrier();
        for( size_t j = 0; j < REPETITIONS; j++)
            connection.post_receive(receive_buffer);
#endif
#ifdef READ
        *send_memory = (char)0;
        *receive_memory = (char)1;
#endif

        connection.barrier();
        auto start_time = TimeTools::now();

            for (size_t i = 0; i < REPETITIONS; i++) {
                if(rank == 0) {

#ifdef WRITE
                    *send_memory = (char)1;
                    connection.write(send_buffer, remote_receive_token.get());
                    while(((volatile char*)receive_memory)[0] != (char)1){}
                    receive_memory[0] = 0;
#endif
#ifdef READ
                    connection.read(send_buffer, remote_receive_token.get(), pRequestToken);
                    while(((volatile char*)send_memory)[0] != (char)1){}
                    send_memory[0] = 0;
#endif
#ifdef SEND
                    connection.send(send_buffer);
                    connection.wait_for_receive(receiveElement);
#endif
                }
                else {
#ifdef WRITE
                    *send_memory = (char)1;
                    while(((volatile char*)receive_memory)[0] != (char)1){}
                    receive_memory[0] = 0;
                    connection.write(send_buffer, remote_receive_token.get());
#endif
#ifdef SEND
                    connection.wait_for_receive(receiveElement);
                    connection.send(send_buffer);
#endif

                }
            }

            if(rank == 0) {
            }
            else {

#ifdef WRITE
#ifdef GPU
                gpu_wait_for_value_and_reset<<<1,1>>>(receive_memory, REPETITIONS-1, 0);
                hipDeviceSynchronize();
#else
                //while(*((volatile char*)receive_memory) != REPETITIONS-1){}
#endif
#endif
            }

        auto end_time = TimeTools::now();

#ifdef READ
        measured_times.push_back(((end_time-start_time)/REPETITIONS));
#else
        measured_times.push_back(((end_time-start_time)/REPETITIONS/2));
#endif
        TRACE("Done with iteration %d!\n", iteration);

    }
    printf("Done with Measurement!\n");
    NodeParameters nodeParameters("",
#ifdef GPU
            true
#else
            false
#endif
            ,false);
    auto parameters = ComputationParameters::generate(connection, nodeParameters);
    BenchmarkTools::output_results(measured_times, output_path, parameters,"VerbsLatency", 1, 1, REPETITIONS);

    delete send_buffer;
    delete receive_buffer;

#ifdef GPU
    hipFree(send_memory);
    hipFree(receive_memory);
#else
    free(send_memory);
    free(receive_memory);
#endif
}

int main(int argc, char** argv) {
    if (argc < 4) {
        std::cerr << "Missing parameter!" <<
                  std::endl;
        std::cerr << "Usage: " << argv[0] << "<NUMA_NODE> <RANK> <Output path>" <<
                  std::endl;
        return -1;
    }

    size_t numa_node = std::stoi( argv[1]);
    pin_to_numa(numa_node);

    size_t rank = static_cast<size_t>( std::stoi( argv[2] ));

    MPIHelper::set_rank(rank);
    MPIHelper::set_process_count(2);
    std::string result_path = argv[3];
    run_test(result_path, numa_node);
}