#include "hip/hip_runtime.h"
//
// Created by Toso, Lorenzo on 2019-01-15.
//

/*
 * This Source Code Form is subject to the terms of the Mozilla Public License,
 * v. 2.0. If a copy of the MPL was not distributed with this file, You can
 * obtain one at http://mozilla.org/MPL/2.0/.
 *
 *
 * Copyright 2018 German Research Center for Artificial Intelligence (DFKI)
 * Author: Clemens Lutz <clemens.lutz@dfki.de>
 *
 * Note that parts of this code are based on the Hawk query compiler by
 * Sebastian Breß et al.
 * See https://github.com/TU-Berlin-DIMA/Hawk-VLDBJ for details.
 */


/*
 * Assumptions:
 *
 * Hash table's size is 2^x - 1
 * Key with value==-1 is reserved for NULL values
 * Key and payload are int64_t
 * Hash table is initialized with all entries set to -1
 *
 * TODO:
 * - use HtEntry struct for key/payload pairs and adjust insert/probe logic
 */

/* See Richter et al., Seven-Dimensional Analysis of Hashing Methods
 * Multiply-shift hash function
 * Requirement: hash factor is an odd 64-bit integer
*/
#include "operators.h"
#include <cstddef>
#include <stdio.h>

#define HASH_FACTOR 123456789123456789ull

#define NULL_KEY 0xFFFFFFFFFFFFFFFFll


__device__
void gpu_insert_tuple(
        int64_t* __restrict__ hash_table, // 128* -1
        size_t hash_table_mask,           // 127
        int64_t key,                        // key
        int64_t payload                     // 0
)
{
    uint32_t index = static_cast<uint32_t>(key);
    index *= HASH_FACTOR;

    for (uint32_t i = 0; i < hash_table_mask + 1; ++i, index += 2) {
        // Effectively index = index % ht_size
        index &= hash_table_mask;

        // Effectively a index -= index % 2
        // This is done because each key/payload pair occupies 2 slots in ht array
        index &= ~1ul;

        unsigned long long int null_key = NULL_KEY;
        int64_t old = hash_table[index];
        if (old == NULL_KEY) {
            old = (int64_t)atomicCAS((unsigned long long int*)&hash_table[index], null_key, (unsigned long long int)key);
            if (old == NULL_KEY) {
                hash_table[index + 1] = payload;
                return;
            }
        }
    }
}


__global__
void gpu_build_hash_map(
        size_t num_elements,
        const int64_t* const __restrict__ join_column_data,
        size_t hash_table_size,
        int64_t* __restrict__ hash_table,
        size_t index_offset
)
{
    const uint32_t global_tid = blockIdx.x *blockDim.x + threadIdx.x;
    const uint32_t number_of_threads = blockDim.x * gridDim.x;


    for(uint32_t tuple_index = global_tid; tuple_index < num_elements; tuple_index += number_of_threads)
    {
        gpu_insert_tuple(
                hash_table,
                hash_table_size - 1,
                join_column_data[tuple_index],
                tuple_index + index_offset
        );
    }
}

__device__
bool gpu_probe_tuple(
        int64_t const* const __restrict__ hash_table,
        size_t hash_table_mask,
        int64_t search_key,
        int64_t* found_payload
)
{
    uint32_t index = static_cast<uint32_t>(search_key);
    index *= HASH_FACTOR;

    for (uint32_t i = 0; i < hash_table_mask + 1; ++i, index += 2) {
        index &= hash_table_mask;
        index &= ~1ul;


        if (hash_table[index] == search_key) {
            *found_payload = hash_table[index + 1];
            return true;
        } else if (hash_table[index] == NULL_KEY) {
            return false;
        }
    }

    return false;
}


__global__
void gpu_probe(
        int64_t num_elements,
        const int64_t* const __restrict__ join_attribute_data,
        int64_t hash_table_length,
        const int64_t* const __restrict__ hash_table,
        int64_t* __restrict__ aggregation_result
)
{
    const uint32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t number_of_threads = blockDim.x * gridDim.x;

    for(uint32_t tuple_index = global_idx; tuple_index < num_elements; tuple_index += number_of_threads)
    {
        int64_t hash_table_payload;
        if (
                gpu_probe_tuple(
                        hash_table,
                        hash_table_length - 1,
                        join_attribute_data[tuple_index],
                        &hash_table_payload)
                )
        {
            aggregation_result[tuple_index] = hash_table_payload;
        } else {
            aggregation_result[tuple_index] = -1;
        }
    }
}

extern __global__
void gpu_count_matches(
        int64_t num_elements,
        const int64_t* const __restrict__ results,
        unsigned int * matches
)
{
    const uint32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t number_of_threads = blockDim.x * gridDim.x;

    uint32_t local_sum = 0;

    for(uint32_t tuple_index = global_idx; tuple_index < num_elements; tuple_index += number_of_threads)
    {
        if(results[tuple_index] != -1 )
            local_sum++;
    }
    atomicAdd(matches, local_sum);

}