#include "hip/hip_runtime.h"
#include <Util/GPUKernnelWrapper/SimpleKernel.cuh>

__global__ void simpleAdditionKernel(const InputRecord* recordValue, const int64_t count, InputRecord* result) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < count) {
        result[i].test$id = recordValue[i].test$id;
        result[i].test$one = recordValue[i].test$one;
        result[i].test$value = recordValue[i].test$value + 42;
    }
}

void SimpleKernelWrapper::execute(int64_t numberOfTuple, InputRecord* d_record, InputRecord* d_result) {
    // prepare kernel launch configuration
    dim3 dimBlock(1024, 1, 1);// using 1D kernel, vx * vy * vz must be <= 1024
    dim3 dimGrid((numberOfTuple + dimBlock.x - 1) / dimBlock.x, 1, 1);

    // launch the kernel
    simpleAdditionKernel<<<dimGrid, dimBlock>>>(d_record, numberOfTuple, d_result);
}
