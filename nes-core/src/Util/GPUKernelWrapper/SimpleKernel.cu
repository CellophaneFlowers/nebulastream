#include "hip/hip_runtime.h"
#include <Util/GPUKernnelWrapper/SimpleKernel.cuh>

__global__ void simpleAdditionKernel(const InputRecord* recordValue, const int64_t count, InputRecord* result) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < count) {
        result[i].test$id = recordValue[i].test$id;
        result[i].test$one = recordValue[i].test$one;
        result[i].test$value = recordValue[i].test$value + 42;
    }
}

void SimpleKernelWrapper::execute(int64_t numberOfTuple, InputRecord* record) {
    // allocate GPU memory to work with the record
    InputRecord* d_record;
    hipMalloc(&d_record, numberOfTuple * sizeof(InputRecord));

    // copy the record to the GPU memory
    hipMemcpy(d_record, record, numberOfTuple * sizeof(InputRecord), hipMemcpyHostToDevice);

    // prepare kernel launch configuration
    dim3 dimBlock(1024, 1, 1);// using 1D kernel, vx * vy * vz must be <= 1024
    dim3 dimGrid((numberOfTuple + dimBlock.x - 1) / dimBlock.x, 1, 1);

    // allocate GPU memory to store the result
    InputRecord* d_result;
    hipMalloc(&d_result, numberOfTuple * sizeof(InputRecord));

    // launch the kernel
    simpleAdditionKernel<<<dimGrid, dimBlock>>>(d_record, numberOfTuple, d_result);

    // copy the result back to host record
    hipMemcpy(record, d_result, numberOfTuple * sizeof(InputRecord), hipMemcpyDeviceToHost);
}
